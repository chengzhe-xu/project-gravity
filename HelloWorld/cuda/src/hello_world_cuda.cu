
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world_cuda() {
    if (blockIdx.x==0 && threadIdx.x==0) printf("Hello world from GPU.\n");
    return;
}

int main() {
    hello_world_cuda<<<16, 32>>>();
    hipDeviceSynchronize();
    return 0;
}