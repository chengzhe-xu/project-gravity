#include "hip/hip_runtime.h"
#include "matrix_mul.h"
#include "cuda_utils.h"

__global__ void matrix_mul_smit_kernel_32x32(float* matA, float* matBT, float* matC, int M, int N, int K) {
    // TODO
    return;
}

matrix_template matrix_mul_smit_host(const matrix_template& matA, const matrix_template& matBT, matrix_template& matC, int M, int N, int K) {
    event_pair timer;
    // hipMalloc device arrays
    float* device_matA = 0;
    float* device_matBT = 0;
    float* device_matC = 0;
    hipMalloc((void**)&device_matA, M * K * sizeof(float));
    hipMalloc((void**)&device_matBT, N * K * sizeof(float));
    hipMalloc((void**)&device_matC, M * N * sizeof(float));
    if(device_matA == 0 || device_matBT == 0 || device_matC == 0) {
        printf("couldn't allocate memory\n");
        return matC;
    }
    // cuda mem copy
    hipMemcpy(device_matA, matA.data(), M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_matBT, matBT.data(), N * K * sizeof(float), hipMemcpyHostToDevice);
    // kernel call
    // note that the size is 2048 * 512, we choose 32 * 32 kernels
    int block_size = 32 * 32;
    int grid_size = (M * N) / block_size;
    start_timer(&timer);
    matrix_mul_smit_kernel_32x32<<<grid_size, block_size>>>(device_matA, device_matBT, device_matC, M, N, K);
    float kernel_time_ms = stop_timer(&timer);
    hipMemcpy(matC.data(), device_matC, M * N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_matA);
    hipFree(device_matBT);
    hipFree(device_matC);
    printf("cuda kernel <matrix_mul_smit_host> runtime %f ms.\n", kernel_time_ms);
    return matC;
}
