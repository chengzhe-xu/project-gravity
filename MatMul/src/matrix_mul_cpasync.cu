#include "hip/hip_runtime.h"
#include "matrix_mul.cuh"
#include "cuda_utils.h"

#define LDG2S(a_share, b_share) \
{ \
    __half2 tmp_a[4], tmp_b[4]; \
    ldg128(from_a, tmp_a[0], tmp_a[1], tmp_a[2], tmp_a[3]); \
    ldg128(from_b, tmp_b[0], tmp_b[1], tmp_b[2], tmp_b[3]); \
    _Pragma("unroll") \
    for (int i=0; i<4; ++i){ \
        (a_share+to_As+i*2*(128+LD_buffer))[0] = tmp_a[i].x; \
        (a_share+to_As+(i*2+1)*(128+LD_buffer))[0] = tmp_a[i].y; \
    } \
    _Pragma("unroll") \
    for (int i=0; i<4; ++i) { \
        (b_share+to_Bs+i*2)[0] = tmp_b[i].x; \
        (b_share+to_Bs+(i*2+1))[0] = tmp_b[i].y; \
    } \
    from_a += 8; from_b += (16*N)/2; \
} \

#define MATMUL_WMMA(a_share, b_share) \
{ \
    nvcuda::wmma::load_matrix_sync(frag_a[0], a_share+warp_row*32, 128+LD_buffer); \
    nvcuda::wmma::load_matrix_sync(frag_a[1], a_share+warp_row*32+16, 128+LD_buffer); \
    _Pragma("unroll") \
    for (int i=0; i<4; ++i) { \
        nvcuda::wmma::load_matrix_sync(frag_b[i], b_share+warp_col*64+16*i, 128+LD_buffer); \
        nvcuda::wmma::mma_sync(frag_acc[0][i], frag_a[0], frag_b[i], frag_acc[0][i]); \
        nvcuda::wmma::mma_sync(frag_acc[1][i], frag_a[1], frag_b[i], frag_acc[1][i]); \
    } \
} \

// #define __HALF2_TO_UI(var) *(reinterpret_cast<unsigned int *>(&(var))) from cuda_fp16.hpp
__device__ __forceinline__ void ldg64(const __half2* addr, __half2 &reg0, __half2 &reg1){
    unsigned int reg0_ui, reg1_ui;
    asm volatile(
        "ld.global.nc.v2.b32 {%0, %1}, [%2];\n"
        : "=r"(reg0_ui),
          "=r"(reg1_ui)
        : "l"(addr)
    );
    reg0 = *(reinterpret_cast<__half2 *>(&reg0_ui));
    reg1 = *(reinterpret_cast<__half2 *>(&reg1_ui));
}

__device__ __forceinline__ void ldg128(const __half2* addr, __half2 &reg0, __half2 &reg1, __half2 &reg2, __half2 &reg3){
    unsigned int reg0_ui, reg1_ui, reg2_ui, reg3_ui;
    asm volatile(
        "ld.global.nc.v4.b32 {%0, %1, %2, %3}, [%4];\n"
        : "=r"(reg0_ui),
          "=r"(reg1_ui),
          "=r"(reg2_ui),
          "=r"(reg3_ui)
        : "l"(addr)
    );
    reg0 = *(reinterpret_cast<__half2 *>(&reg0_ui));
    reg1 = *(reinterpret_cast<__half2 *>(&reg1_ui));
    reg2 = *(reinterpret_cast<__half2 *>(&reg2_ui));
    reg3 = *(reinterpret_cast<__half2 *>(&reg3_ui));
}

__device__ __forceinline__ void sts32(const __half* addr, __half &reg0, __half &reg1){
    __half2* addr_shared_state = reinterpret_cast<__half2 *>(__cvta_generic_to_shared(addr));
    asm volatile(
        "st.shared.v2.b16 [%0], {%1, %2};\n"
        :
        : "l"(addr_shared_state),
          "h"(*(reinterpret_cast<unsigned short *>(&reg0))),
          "h"(*(reinterpret_cast<unsigned short *>(&reg1)))
    );
}

__device__ __forceinline__ void stg128(__half2* addr, __half2 &reg0, __half2 &reg1, __half2 &reg2, __half2 &reg3) {
    asm volatile(
        "st.global.v4.b32 [%0], {%1, %2, %3, %4};\n"
        :
        : "l"(addr),
          "r"(*(reinterpret_cast<unsigned int *>(&reg0))),
          "r"(*(reinterpret_cast<unsigned int *>(&reg1))),
          "r"(*(reinterpret_cast<unsigned int *>(&reg2))),
          "r"(*(reinterpret_cast<unsigned int *>(&reg3)))
    );
}

// __device__ __forceinline__ void ldgsts32(__half2* shared_addr, __half2* global_addr, bool guard) {
//     __half2* addr_shared_state = reinterpret_cast<__half2 *>(__cvta_generic_to_shared(shared_addr));
//     asm volatile(
//         "cp.async.ca.shared.global [%0], [%1], 4;}\n"
//         :
//         : "l"(addr_shared_state), 
//           "l"(global_addr)
//     );
// }

/*
This implementation is the SIMT core version.
For each block, we assign 16*16 threads,
For each thread, we assign 8*8 C matrix
For each block, we assign 128*128 C matrix,
For each warp, we assign 32*64 C matrix
For each step, we set k = 16
*/

/*
to debug matrix mul, it is very useful to use "cycle matrix" as input, 
0 1 2 3 4 5 6 7 8 9 0 1
2 3 4 5 6 7 8 9 0 1 2 3
.....
.....
4 5 6 7 8 9 0 1 2 3 4 5
and print out every details in the matmul process, 
mem bias, mem content, pA, pB, and check whether or not the output match expectation
*/

__global__ void matrix_mul_cpasync_kernel_128x128(__half2* matA, __half2* matB, __half2* matC, int M, int N, int K) {
    const unsigned int block_id = blockIdx.x;
    const unsigned int thread_id = threadIdx.x;
    const unsigned int block_row = block_id / (N/128);
    const unsigned int block_col = block_id % (N/128);
    const unsigned int warp_id = thread_id / 32;
    const unsigned int warp_row = warp_id / 2;
    const unsigned int warp_col = warp_id % 2;
    const unsigned int thread_row = (thread_id % 32) / 8;
    const unsigned int thread_col = (thread_id % 32) % 8;

    using fragA_t = nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, __half, nvcuda::wmma::col_major>;
    using fragB_t = nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, __half, nvcuda::wmma::row_major>;
    using fragAcc_t = nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, __half>;

    fragA_t frag_a[2];
    fragB_t frag_b[4];
    fragAcc_t frag_acc[2][4];
    __half* matC_h = reinterpret_cast<__half *>(matC) + (block_row*128 + warp_row*32) * N + block_col*128 + warp_col*64;
    #pragma unroll
    for (int i=0; i<2; ++i) {
        #pragma unroll
        for (int j=0; j<4; ++j) {
            // nvcuda::wmma::fill_fragment(frag_acc[i][j], __half(0.0));
            nvcuda::wmma::load_matrix_sync(frag_acc[i][j], matC_h + i*16*N + j*16, N, nvcuda::wmma::mem_row_major);
        }
    }

    const unsigned int LD_buffer = 16;

    // shared memory
    __shared__ __align__(4 * 1024) char smem[20 * 1024];
    // As/Bs needs 128 * 16 * half = 128 * 16 * 16 bits = 32768 bits = 32768 / 8 char = 4096 char
    // add the LD_buffer: need 4352 char = 4.25 k ==> 4.5 k
    // Cs needs 128 * (128 + LD_buffer) * half = 36864 = 36 k --- now we do not use share memory as a intermedia for acc
    __half* As[2] = {reinterpret_cast<__half *>(smem),
                    reinterpret_cast<__half *>(smem + 5120)};
    __half* Bs[2] = {reinterpret_cast<__half *>(smem + 5120*2),
                    reinterpret_cast<__half *>(smem + 5120*3)};
    // TODO: what is the __align__ used for and why we add some buffer into the share memory?

    // set the outer for loop initial value
    __half2* from_a = matA + (block_row*128 + (thread_id/2)) * (K/2) + 4*(thread_id%2);
    __half2* from_b = matB + (thread_id/16) * (N/2) + block_col*(128/2) + 4*(thread_id%16); 
    unsigned int to_As = (thread_id%2) * 8 * (128+LD_buffer) + (thread_id/2);
    unsigned int to_Bs = (thread_id/16) * (128+LD_buffer) + 8 * (thread_id%16);
    // outer loop
    LDG2S(As[0], Bs[0])
    unsigned int pipeline_indicator = 0;
    #pragma unroll
    for (int i_step=0; i_step<K/16-1; ++i_step) {
        // load sub A, B matrix
        __syncthreads();
        LDG2S(As[1-pipeline_indicator], Bs[1-pipeline_indicator])
        MATMUL_WMMA(As[pipeline_indicator], Bs[pipeline_indicator])
        pipeline_indicator = 1 - pipeline_indicator;
    }
    __syncthreads();
    MATMUL_WMMA(As[pipeline_indicator], Bs[pipeline_indicator])
    #pragma unroll
    for (int i=0; i<2; ++i) {
        #pragma unroll
        for (int j=0; j<4; ++j) {
            nvcuda::wmma::store_matrix_sync(matC_h + i*16*N + j*16, frag_acc[i][j], N, nvcuda::wmma::mem_row_major);
        }
    }
    __syncthreads();
    return;
}

matrix_template matrix_mul_cpasync_host(const matrix_template& matA, const matrix_template& matB, matrix_template& matC, int M, int N, int K) {
    event_pair timer;
    // hipMalloc device arrays
    float* device_matA = 0;
    float* device_matB = 0;
    float* device_matC = 0;
    hipMalloc((void**)&device_matA, M * K * sizeof(float));
    hipMalloc((void**)&device_matB, K * N * sizeof(float));
    hipMalloc((void**)&device_matC, M * N * sizeof(float));
    if(device_matA == 0 || device_matB == 0 || device_matC == 0) {
        printf("couldn't allocate memory\n");
        return matC;
    }
    // __half_copy
    __half* device_matA_h = 0;
    __half* device_matB_h = 0;
    __half* device_matC_h = 0;
    hipMalloc((void**)&device_matA_h, M * K * sizeof(__half));
    hipMalloc((void**)&device_matB_h, K * N * sizeof(__half));
    hipMalloc((void**)&device_matC_h, M * N * sizeof(__half));
    if(device_matA_h == 0 || device_matB_h == 0 || device_matC_h == 0) {
        printf("couldn't allocate memory\n");
        return matC;
    }
    // cuda mem copy
    hipMemcpy(device_matA, matA.data(), M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_matB, matB.data(), K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_matC, matC.data(), M * N * sizeof(float), hipMemcpyHostToDevice);

    cast_kernel_float2half<<<128, 256>>>(device_matA_h, device_matA, M * K);
    cast_kernel_float2half<<<128, 256>>>(device_matB_h, device_matB, K * N);
    cast_kernel_float2half<<<128, 256>>>(device_matC_h, device_matC, M * N);

    __half2* device_matA_h2 = reinterpret_cast<__half2 *>(device_matA_h); 
    __half2* device_matB_h2 = reinterpret_cast<__half2 *>(device_matB_h);
    __half2* device_matC_h2 = reinterpret_cast<__half2 *>(device_matC_h);

    // kernel call
    int block_size = 16 * 16;
    int grid_size = (M * N) / (128 * 128);
    start_timer(&timer);
    matrix_mul_cpasync_kernel_128x128<<<grid_size, block_size>>>(device_matA_h2, device_matB_h2, device_matC_h2, M, N, K);
    float kernel_time_ms = stop_timer(&timer);
    device_matC_h = reinterpret_cast<__half *>(device_matC_h2);
    cast_kernel_half2float<<<128, 256>>>(device_matC, device_matC_h, M * N);
    hipMemcpy(matC.data(), device_matC, M * N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_matA);
    hipFree(device_matB);
    hipFree(device_matC);
    hipFree(device_matA_h);
    hipFree(device_matB_h);
    hipFree(device_matC_h);
    printf("cuda kernel <matrix_mul_cpasync_kernel_128x128> runtime %f ms.\n", kernel_time_ms);
    return matC;
}
