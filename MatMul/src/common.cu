#include "hip/hip_runtime.h"
#include "matrix_mul.cuh"

__global__ void cast_kernel_float2half(__half* arr_h, float* arr, const int arr_size) {
    const unsigned int stride = gridDim.x * blockDim.x;
    const unsigned int start_idx = blockIdx.x * blockDim.x + threadIdx.x;
    #pragma unroll
    for (int i=start_idx; i<arr_size; i+=stride) {
        arr_h[i] = __float2half(arr[i]);
    }
    __syncthreads();
    return;
}

__global__ void cast_kernel_half2float(float* arr, __half* arr_h, const int arr_size) {
    const unsigned int stride = gridDim.x * blockDim.x;
    const unsigned int start_idx = blockIdx.x * blockDim.x + threadIdx.x;
    #pragma unroll
    for (int i=start_idx; i<arr_size; i+=stride) {
        arr[i] = __half2float(arr_h[i]);
    }
    __syncthreads();
    return;
}

__device__ __forceinline__ void ldg128(const __half2* addr, __half2 &reg0, __half2 &reg1, __half2 &reg2, __half2 &reg3){
    asm volatile(
        "ld.global.nc.v4.b32 {%1, %2, %3, %4}, [%0];\n"
        : "=r"(__HALF2_TO_UI(reg0)),
          "=r"(__HALF2_TO_UI(reg1)),
          "=r"(__HALF2_TO_UI(reg2)),
          "=r"(__HALF2_TO_UI(reg3))
        : "l"(addr)
    );
}

__device__ __forceinline__ void stg128(__half2* addr, __half2 &reg0, __half2 &reg1, __half2 &reg2, __half2 &reg3) {
    asm volatile(
        "st.global.v4.b32 [%0], {%1, %2, %3, %4};\n"
        :
        : "l"(addr),
          "r"(__HALF2_TO_UI(reg0)),
          "r"(__HALF2_TO_UI(reg1)),
          "r"(__HALF2_TO_UI(reg2)),
          "r"(__HALF2_TO_UI(reg3))
    );
}

__device__ __forceinline__ void half2matmulacc(__half2 acc[8][4], __half2 pA[8], __half2 pB[8]) {
    // TODO
    return
}